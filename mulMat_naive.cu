#include "hip/hip_runtime.h"
//matrix multiple (nx*nk)(nk*ny) = (nx*ny)

#include "include/common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "src/utils.c"

const int TILE_SIZE = 16;

__global__ void warmup(Mat* A, Mat* B, Mat* C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;	//结果矩阵C的行索引
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < C->height && col < C->width){	//结果矩阵C的列索引
        for (int e = 0; e < A->width; ++e)
        {
            Cvalue += A->elements[row * A->width + e]			//所有点到点的元素乘积求和
                    * B->elements[e * B->width + col];
        }
	    C->elements[row * C->width + col] = Cvalue;

    }
}


__global__ void MatMulKernel(Mat* A, Mat* B, Mat* C)
{
	// Each thread computes one element of C
	// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;	//结果矩阵C的行索引
	int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < C->height && col < C->width){	//结果矩阵C的列索引
        for (int e = 0; e < A->width; ++e)
        {
            Cvalue += A->elements[row * A->width + e]			//所有点到点的元素乘积求和
                    * B->elements[e * B->width + col];
        }
	    C->elements[row * C->width + col] = Cvalue;

    }
}

int main(int argc, char *argv[]){
    /*
    A: X * K
    B: K * Y
    C, D: X * Y
    */
    Mat *A, *B, *kernel1, *kernel2;
    Mat *Host = (Mat*)malloc(sizeof(Mat));
    CHECK(hipMallocManaged((float **)&A, sizeof(Mat)));
    CHECK(hipMallocManaged((float **)&B, sizeof(Mat)));
    CHECK(hipMallocManaged((float **)&kernel1, sizeof(Mat)));
    CHECK(hipMallocManaged((float **)&kernel2, sizeof(Mat)));

    
    int nshift = 512;
    if (argc > 1){nshift = atoi(argv[1]);}

    int nx, ny, nk;
    nx = ny = nk = nshift;
    A->height = nx;A->width = nk;
    B->height = nk;B->width = ny;
    Host->height = nx;Host->width = ny;
    kernel1->height = nx;kernel1->width = ny;
    kernel2->height = nx;kernel2->width = ny;

    int nxk = nx * nk;
    int nky = nk * ny;
    int nxy = nx *ny;
    int nBytes = sizeof(float) * nxy;
    CHECK(hipMallocManaged((float **)&A->elements, nxk * sizeof(float)));
    CHECK(hipMallocManaged((float **)&B->elements, nky * sizeof(float)));
    CHECK(hipMallocManaged((float **)&kernel1->elements, nxy * sizeof(float)));
    CHECK(hipMallocManaged((float **)&kernel2->elements, nxy * sizeof(float)));


    Host->elements = (float *)malloc(nBytes);

    double iStart = seconds();
    initdata(A->elements, nxk);
    initdata(B->elements, nky);
    double iElaps = seconds() - iStart;
    // printf("initialization: \t%f\n", iElaps);

    dim3 block(TILE_SIZE, TILE_SIZE);
    dim3 grid((nx + block.x - 1)/block.x, (ny + block.y - 1)/block.y);
    iStart = seconds();
    warmup<<<grid, block>>>(A, B, kernel1);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    //printf("mulMat on warmup: \t%f\n", iElaps);

    iStart = seconds();
    MatMulKernel<<<grid, block>>>(A, B, kernel2);
    CHECK(hipDeviceSynchronize());
    iElaps = seconds() - iStart;
    printf("mulMat on device orgin: \t%f\n", iElaps);
    CHECK(hipGetLastError());
    checkResult(kernel2, kernel1);
    return 0;
}